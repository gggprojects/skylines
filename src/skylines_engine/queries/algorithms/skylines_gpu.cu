#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>

#include "gpu/gpu_memory.hpp"
#include "queries/data/data_structures.hpp"
#include "queries/algorithms/algorithm.cuh"

__constant__ sl::queries::data::Point device_input_q[8192];

#define SHARED_MEM_SIZE 1024

__global__ void ComputePartialSkyline(
    const sl::queries::data::WeightedPoint *input_p, 
    size_t input_p_size,
    int input_q_size,
    unsigned int *result) {

    __shared__ sl::queries::data::WeightedPoint shared_input_p[SHARED_MEM_SIZE];

    int block_offset = blockIdx.x * blockDim.x; // we just have one dimension grids
    size_t global_pos = block_offset + threadIdx.x;

    sl::queries::data::WeightedPoint skyline_candidate(input_p[global_pos]);
    bool is_skyline = global_pos < input_p_size;

    for (size_t current_input_p_pos = 0; current_input_p_pos < input_p_size; current_input_p_pos += SHARED_MEM_SIZE) {
        //all threads in the block loads to shared
        shared_input_p[threadIdx.x] = input_p[threadIdx.x + current_input_p_pos];
        __syncthreads();

        if (is_skyline) {
            #pragma unroll SHARED_MEM_SIZE
            for (int i = 0; i < SHARED_MEM_SIZE; i++) {
                if (current_input_p_pos + i != global_pos) { // do not check against the same point
                    if (IsDominated_impl(skyline_candidate, shared_input_p[i], device_input_q, input_q_size)) {
                        is_skyline = false;
                        break;
                    }
                }
            }
        }
        __syncthreads();
    }

    result[global_pos] = is_skyline ? 1 : 0;
}

template<typename T>
T inline divUp(T a, T b) {
    return (a + b - 1) / b;
}

template<typename T>
T roundUp(T numToRound, T multiple)
{
    if (multiple == 0)
        return numToRound;

    T remainder = numToRound % multiple;
    if (remainder == 0)
        return numToRound;

    return numToRound + multiple - remainder;
}

extern "C" void ComputeGPUSkyline(
    const std::vector<sl::queries::data::WeightedPoint> &input_p,
    const std::vector<sl::queries::data::Point> &input_q,
    std::vector<sl::queries::data::WeightedPoint> *output) {

    sl::gpu::GPUStream gpu_stream;

    //copy to const memory the input Q
    hipMemcpyToSymbolAsync(HIP_SYMBOL(device_input_q), input_q.data(), sizeof(sl::queries::data::Point) * input_q.size(), 0, hipMemcpyKind::hipMemcpyHostToDevice, gpu_stream());

    size_t input_p_size = input_p.size();
    int input_q_size = static_cast<int>(input_q.size());

    size_t input_p_size_SHARED_MEM_SIZE_multiple = roundUp<size_t>(input_p.size(), SHARED_MEM_SIZE);

    //copy to global memory the input P
    sl::gpu::GPUMemory<sl::queries::data::WeightedPoint> input_p_d(input_p_size_SHARED_MEM_SIZE_multiple);
    input_p_d.UploadToDeviceAsync(input_p, gpu_stream); //the final values maybe empty

    size_t remaining_positions = input_p_size_SHARED_MEM_SIZE_multiple - input_p_size;
    std::vector<sl::queries::data::WeightedPoint> remaining_points(remaining_positions, sl::queries::data::WeightedPoint(sl::queries::data::Point(2., 2.), 1));
    input_p_d.UploadToDeviceAsync(remaining_points, input_p_size, gpu_stream);

    sl::gpu::GPUMemory<unsigned int> result_d(input_p_size_SHARED_MEM_SIZE_multiple);
    /*
    MAX number of threads per MS is 2048.
    MAX number of threads per block 1024 => max blockDim.y = 1
    */
    dim3 threadsPerBlock(SHARED_MEM_SIZE, 1);
    int total_numBlocks = static_cast<int>(divUp(input_p_size, static_cast<size_t>(threadsPerBlock.x * threadsPerBlock.y)));
    dim3 grid(total_numBlocks, 1);

    ComputePartialSkyline<<< grid, threadsPerBlock, 0, gpu_stream() >>>(input_p_d(), input_p_size, input_q_size, result_d());

    std::vector<unsigned int> result(input_p_size);
    result_d.DownloadToHostAsync(result.data(), input_p_size, gpu_stream);
    gpu_stream.Syncronize();

    for (size_t i = 0; i < result.size(); i++) {
        if (result[i] == 1) {
            output->push_back(input_p[i]);
        }
    }
}



