#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <set>

#include <hip/hip_runtime.h>

#include "gpu/gpu_memory.hpp"
#include "queries/data/data_structures.hpp"
#include "queries/algorithms/algorithm.cuh"
#include "queries/algorithms/distance_type.hpp"

#define SHARED_MEM_ELEMENTS 1024

/*
Total amount of constant memory:    65536 bytes
sizeof(sl::queries::data::Point):   8 bytes
Max elements:                       65536 / 8 = 8192
*/
#define MAX_CONST_MEM_ELEMENTS 8192

__constant__ sl::queries::data::Point device_input_q[MAX_CONST_MEM_ELEMENTS];

__device__ inline bool NeartestFunc(const float a, const float b) {
    return a <= b;
}

__device__ inline bool FurthestFunc(const float a, const float b) {
    return a >= b;
}

template<class Comparator>
__device__ void _ComputePartialSkyline(
    const sl::queries::data::WeightedPoint *input_p,
    size_t input_p_size,
    int input_q_size,
    Comparator comparator_function,
    sl::queries::data::Statistics *statistics,
    float *result) {

    __shared__ sl::queries::data::WeightedPoint shared_input_p[SHARED_MEM_ELEMENTS];

    int block_offset = blockIdx.x * blockDim.x; // we just have one dimension grids
    size_t global_pos = block_offset + threadIdx.x;

    sl::queries::data::WeightedPoint skyline_candidate(input_p[global_pos]);
    bool is_skyline = global_pos < input_p_size;

    sl::queries::data::Statistics thread_statistics;
    for (size_t current_input_p_pos = 0; current_input_p_pos < input_p_size; current_input_p_pos += SHARED_MEM_ELEMENTS) {
        //all threads in the block loads to shared
        shared_input_p[threadIdx.x] = input_p[threadIdx.x + current_input_p_pos];
        __syncthreads();

        if (is_skyline) {
            for (int i = 0; i < SHARED_MEM_ELEMENTS; i++) {
                if (current_input_p_pos + i != global_pos && current_input_p_pos + i < input_p_size) { // do not check against the same point
                    if (sl::queries::algorithms::IsDominated(skyline_candidate, shared_input_p[i], device_input_q, input_q_size, comparator_function)) {
                        is_skyline = false;
                        break;
                    }
                    thread_statistics.num_comparisions_++;
                }
            }
        }
        __syncthreads();
    }

    if (is_skyline) {
        float max_distance = 0;
        for (int i = 0; i < input_q_size; i++) {
            float distance = skyline_candidate.SquaredDistance(device_input_q[i]);
            if (distance > max_distance) {
                max_distance = distance;
            }
        }
        result[global_pos] = max_distance;
    } else {
        result[global_pos] = -1;
    }

    atomicAdd(&statistics->num_comparisions_, thread_statistics.num_comparisions_);
}

__global__ void ComputePartialSkyline(
    const sl::queries::data::WeightedPoint *input_p, 
    size_t input_p_size,
    int input_q_size,
    sl::queries::algorithms::DistanceType distance_type,
    sl::queries::data::Statistics *statistics,
    float *result) {

    switch (distance_type) {
        case sl::queries::algorithms::DistanceType::Neartest:
            _ComputePartialSkyline(input_p, input_p_size, input_q_size, NeartestFunc, statistics, result);
            break;
        case sl::queries::algorithms::DistanceType::Furthest:
            _ComputePartialSkyline(input_p, input_p_size, input_q_size, FurthestFunc, statistics, result);
            break;
        default:
            break;
    }
}

template<typename T>
T inline divUp(T a, T b) {
    return (a + b - 1) / b;
}

template<typename T>
T roundUp(T numToRound, T multiple)
{
    if (multiple == 0)
        return numToRound;

    T remainder = numToRound % multiple;
    if (remainder == 0)
        return numToRound;

    return numToRound + multiple - remainder;
}

extern "C" bool CheckInputCorrectness(const std::vector<sl::queries::data::WeightedPoint> &input_p,
    const std::vector<sl::queries::data::Point> &input_q) {
    if (input_q.size() > MAX_CONST_MEM_ELEMENTS) return false;
    return true;
}

extern "C" void ComputeGPUSkyline(
    const std::vector<sl::queries::data::WeightedPoint> &input_p,
    const std::vector<sl::queries::data::Point> &input_q,
    std::vector<sl::queries::data::WeightedPoint> *output,
    sl::queries::algorithms::DistanceType distance_type,
    size_t top_k,
    sl::queries::data::Statistics *stadistics_results) {

    sl::gpu::GPUStream gpu_stream;

    //copy to const memory the input Q
    hipMemcpyToSymbolAsync(HIP_SYMBOL(device_input_q), input_q.data(), sizeof(sl::queries::data::Point) * input_q.size(), 0, hipMemcpyKind::hipMemcpyHostToDevice, gpu_stream());

    size_t input_p_size = input_p.size();
    int input_q_size = static_cast<int>(input_q.size());

    size_t input_p_size_SHARED_MEM_SIZE_multiple = roundUp<size_t>(input_p.size(), SHARED_MEM_ELEMENTS);

    //copy to global memory the input P
    sl::gpu::GPUMemory<sl::queries::data::WeightedPoint> input_p_d(input_p_size_SHARED_MEM_SIZE_multiple);
    input_p_d.UploadToDeviceAsync(input_p, gpu_stream); //the final values maybe empty

    //copy statistics
    sl::gpu::GPUMemory<sl::queries::data::Statistics> statistics_d(1);
    statistics_d.UploadToDeviceAsync(stadistics_results, 1, gpu_stream);

    sl::gpu::GPUMemory<float> result_d(input_p_size_SHARED_MEM_SIZE_multiple);
    /*
    MAX number of threads per MS is 2048.
    MAX number of threads per block 1024 => max blockDim.y = 1
    */
    dim3 threadsPerBlock(SHARED_MEM_ELEMENTS, 1);
    int total_numBlocks = static_cast<int>(divUp(input_p_size, static_cast<size_t>(threadsPerBlock.x * threadsPerBlock.y)));
    dim3 grid(total_numBlocks, 1);

    ComputePartialSkyline<<< grid, threadsPerBlock, 0, gpu_stream() >>> (input_p_d(), input_p_size, input_q_size, distance_type, statistics_d(), result_d());
    std::vector<float> result(input_p_size);
    result_d.DownloadToHostAsync(result.data(), input_p_size, gpu_stream);
    statistics_d.DownloadToHostAsync(stadistics_results, gpu_stream);

    gpu_stream.Syncronize();

    std::set<sl::queries::algorithms::PointStatistics> points;
    float max_distance_in_set = 99999;
    for (size_t i = 0; i < result.size(); i++) {
        float distance = result[i];
        if (distance != -1) {
            //it's a skyline
            if (points.size() < top_k || distance < max_distance_in_set) {
                points.insert(sl::queries::algorithms::PointStatistics(input_p[i], std::make_pair(0.f, distance)));
                if (points.size() > top_k)
                    points.erase(points.begin());
                max_distance_in_set = points.begin()->s_.second;
            }
        }
    }

    for (const sl::queries::algorithms::PointStatistics &ps : points) {
        output->emplace_back(ps.wp_);
    }
}



